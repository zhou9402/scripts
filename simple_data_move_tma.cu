#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>

#define CUDA_CHECK(call)                                                                                               \
    do                                                                                                                 \
    {                                                                                                                  \
        hipError_t error = call;                                                                                      \
        if (error != hipSuccess)                                                                                      \
        {                                                                                                              \
            fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\"\n", __FILE__, __LINE__, error, hipGetErrorName(error), hipGetErrorString(error)); \
            exit(EXIT_FAILURE);                                                                                        \
        }                                                                                                              \
    } while (0)



__device__ __forceinline__ void fence_view_async_shared() {
    asm volatile("fence.proxy.async.shared::cta; \n" :: );
}

__device__ __forceinline__ void fence_barrier_init() {
    asm volatile("fence.mbarrier_init.release.cluster; \n" :: );
}

__device__ __forceinline__ void mbarrier_init(uint64_t* mbar_ptr, uint32_t arrive_count) {
    auto mbar_int_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(mbar_ptr));
    asm volatile("mbarrier.init.shared::cta.b64 [%1], %0;" :: "r"(arrive_count), "r"(mbar_int_ptr));
}

__device__ __forceinline__ void mbarrier_wait(uint64_t* mbar_ptr, uint32_t& phase) {
    auto mbar_int_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(mbar_ptr));
    asm volatile("{\n\t"
                 ".reg .pred       P1; \n\t"
                 "LAB_WAIT: \n\t"
                 "mbarrier.try_wait.parity.shared::cta.b64 P1, [%0], %1, %2; \n\t"
                 "@P1 bra DONE; \n\t"
                 "bra     LAB_WAIT; \n\t"
                 "DONE: \n\t"
                 "}" :: "r"(mbar_int_ptr), "r"(phase), "r"(0x989680));
    phase ^= 1;
}

__device__ __forceinline__ void mbarrier_arrive_and_expect_tx(uint64_t* mbar_ptr, int num_bytes) {
    auto mbar_int_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(mbar_ptr));
    asm volatile("mbarrier.arrive.expect_tx.shared::cta.b64 _, [%1], %0; \n\t" :: "r"(num_bytes), "r"(mbar_int_ptr));
}

__device__ __forceinline__ void tma_store_fence() {
    asm volatile ("fence.proxy.async.shared::cta;");
}

constexpr uint64_t kEvictFirst = 0x12f0000000000000;
constexpr uint64_t kEvictNormal = 0x1000000000000000;

__device__ __forceinline__ void tma_load_1d(const void* smem_ptr, const void* gmem_ptr, uint64_t* mbar_ptr, int num_bytes,
                                            bool evict_first = true) {
    auto mbar_int_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(mbar_ptr));
    auto smem_int_ptr  = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
    const auto cache_hint = evict_first ? kEvictFirst : kEvictNormal;
    asm volatile("cp.async.bulk.shared::cluster.global.mbarrier::complete_tx::bytes.L2::cache_hint [%0], [%1], %2, [%3], %4;\n"
                 :: "r"(smem_int_ptr), "l"(gmem_ptr), "r"(num_bytes), "r"(mbar_int_ptr), "l"(cache_hint) : "memory");
}

__device__ __forceinline__ void tma_store_1d(const void* smem_ptr, const void* gmem_ptr, int num_bytes,
                                             bool evict_first = true) {
    auto smem_int_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
    const auto cache_hint = evict_first ? kEvictFirst : kEvictNormal;
    asm volatile("cp.async.bulk.global.shared::cta.bulk_group.L2::cache_hint [%0], [%1], %2, %3;\n"
                 :: "l"(gmem_ptr), "r"(smem_int_ptr), "r"(num_bytes), "l"(cache_hint) : "memory");
    asm volatile("cp.async.bulk.commit_group;");
}

template <int N = 0>
__device__ __forceinline__ void tma_store_wait() {
    asm volatile("cp.async.bulk.wait_group.read %0;" :: "n"(N) : "memory");
}


// A proper TMA bulk copy kernel for Hopper - Grid-stride loop with 32KB per iteration
__global__ void tma_bulk_copy_kernel(
    void *__restrict__ vdst,
    const void *__restrict__ vsrc,
    int N)
{
    // 每次copy 32KB = 8192个float
    const int elements_per_chunk = 8192;  // 32KB / 4 bytes per float
    const int bytes_per_chunk = elements_per_chunk * sizeof(float);  // 32KB
    
    extern __shared__ __align__(1024) uint8_t smem_buffer[];
    auto *dst = reinterpret_cast<float *>(vdst);
    const auto *src = reinterpret_cast<const float *>(vsrc);
    
    // TMA barrier for load
    __shared__ uint64_t tma_mbarrier;
    uint32_t tma_phase = 0;

    if (threadIdx.x == 0) {
        mbarrier_init(&tma_mbarrier, 1);
        fence_view_async_shared();
        fence_barrier_init();
    }
    __syncthreads();

    // Grid-stride loop: 每个block轮流处理32KB数据块
    for (int chunk_start = blockIdx.x * elements_per_chunk; 
         chunk_start < N; 
         chunk_start += gridDim.x * elements_per_chunk) {
        
        // 计算当前chunk的实际大小
        int chunk_end = min(chunk_start + elements_per_chunk, N);
        int elements_to_process = chunk_end - chunk_start;
        int bytes_to_process = elements_to_process * sizeof(float);
        
        // 跳过空的chunk
        if (elements_to_process <= 0) {
            break;
        }

        // --- TMA Load: Global -> Shared ---
        if (threadIdx.x == 0)
        {
            // 计算当前chunk的源地址和目标地址
            const float* chunk_src = src + chunk_start;
            float* chunk_dst = dst + chunk_start;
            
            // TMA加载：从global memory的当前chunk位置加载到shared memory
            tma_load_1d(smem_buffer, chunk_src, &tma_mbarrier, bytes_to_process);
            mbarrier_arrive_and_expect_tx(&tma_mbarrier, bytes_to_process);
            
            // 等待TMA加载完成
            mbarrier_wait(&tma_mbarrier, tma_phase);
            
            // --- TMA Store: Shared -> Global ---
            // TMA存储：从shared memory存储到global memory的当前chunk位置
            tma_store_1d(smem_buffer, chunk_dst, bytes_to_process);
            tma_store_wait();
        }
        
        // 等待当前chunk的TMA操作完成再进行下一个chunk
        __syncthreads();
    }
}

int main(int argc, char **argv)
{
    printf("=============================================================================\n");
    printf("TMA (Tensor Memory Accelerator) Grid-Stride Copy Test - 32KB per chunk\n");
    printf("=============================================================================\n");

    
    hipDeviceProp_t prop;
    int device_id = 0;
    CUDA_CHECK(hipGetDevice(&device_id));
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    printf("GPU: %s (compute capability %d.%d)\n", prop.name, prop.major, prop.minor);

    if (prop.major < 9)
    {
        printf("❌ TMA requires compute capability >= 9.0 (Hopper architecture)\n");
        printf("Current GPU does not support TMA.\n");
        return 1;
    }
    printf("✅ TMA support detected\n");

    // Parameters - 默认更大的数据量以测试多轮copy
    int N = argc > 1 ? atoi(argv[1]) : 256 * 1024;  // 默认256K个float = 1MB

    // TMA requires 16-byte alignment for some operations, good practice to align.
    N = (N + 3) & ~3; 
    printf("Total elements: N = %d floats\n", N);

    size_t size = N * sizeof(float);
    printf("Total data size: %.1f KB (%.1f MB)\n", size / 1024.0f, size / 1024.0f / 1024.0f);
    
    // 配置grid：每个block处理32KB chunks
    const int elements_per_chunk = 8192;  // 32KB / 4 bytes = 8192 floats
    const int chunk_size_kb = 32;
    
    // 计算需要的总chunk数量
    int total_chunks = (N + elements_per_chunk - 1) / elements_per_chunk;
    
    // 设置block数量：可以小于总chunk数，利用grid-stride loop
    // 建议设置为SM数量的倍数以获得好的负载均衡
    int num_sms = prop.multiProcessorCount;
    int num_blocks = min(total_chunks, num_sms * 4);  // 每个SM最多4个block
    
    printf("\nConfiguration:\n");
    printf("  Elements per chunk: %d (%dKB)\n", elements_per_chunk, chunk_size_kb);
    printf("  Total chunks needed: %d\n", total_chunks);
    printf("  Number of SMs: %d\n", num_sms);
    printf("  Number of blocks: %d\n", num_blocks);
    printf("  Threads per block: 1024\n");
    printf("  Each block will process ~%.1f chunks on average\n", (float)total_chunks / num_blocks);

    // Allocate memory
    float *h_src, *h_dst;
    void *d_src, *d_dst;

    h_src = (float *)malloc(size);
    h_dst = (float *)malloc(size);
    CUDA_CHECK(hipMalloc(&d_src, size));
    CUDA_CHECK(hipMalloc(&d_dst, size));

    // Initialize data
    for (int i = 0; i < N; i++)
    {
        h_src[i] = (float)i;
    }
    memset(h_dst, 0, size);

    CUDA_CHECK(hipMemcpy(d_src, h_src, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_dst, 0, size));

    // Events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    
    const int num_runs = 50;
    float times[num_runs];

    // Set shared memory size - 每个block需要32KB用于chunk缓存
    size_t smem_size = elements_per_chunk * sizeof(float);  // 32KB
    printf("\nShared memory per block: %zu bytes (%dKB)\n", smem_size, chunk_size_kb);
    
    if (smem_size > (size_t)prop.sharedMemPerBlock) {
        printf("Error: Requested shared memory size %zu bytes is larger than max %d bytes\n", 
               smem_size, prop.sharedMemPerBlock);
        return 1;
    }
    
    // 设置kernel属性
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(tma_bulk_copy_kernel), 
                                   hipFuncAttributeMaxDynamicSharedMemorySize, 
                                   smem_size));
    
    printf("\nRunning %d TMA grid-stride tests...\n", num_runs);

    // Warm-up run
    printf("Warming up...\n");
    tma_bulk_copy_kernel<<<num_blocks, 1024, smem_size>>>(d_dst, d_src, N);
    CUDA_CHECK(hipDeviceSynchronize());

    // Test
    for (int i = 0; i < num_runs; i++)
    {
        CUDA_CHECK(hipEventRecord(start));
        tma_bulk_copy_kernel<<<num_blocks, 1024, smem_size>>>(d_dst, d_src, N);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));

        CUDA_CHECK(hipEventElapsedTime(&times[i], start, stop));
        if (i < 10 || i % 10 == 9)
        {
            printf("Run %3d: %.3f ms\n", i + 1, times[i]);
        }
    }

    // Calculate statistics
    float min_time = times[0];
    float max_time = times[0];
    float total_time = 0.0f;

    for (int i = 0; i < num_runs; i++)
    {
        total_time += times[i];
        if (times[i] < min_time)
            min_time = times[i];
        if (times[i] > max_time)
            max_time = times[i];
    }

    float avg_time = total_time / num_runs;
    
    // Calculate standard deviation
    float variance = 0.0f;
    for (int i = 0; i < num_runs; i++) {
        float diff = times[i] - avg_time;
        variance += diff * diff;
    }
    float std_dev = sqrt(variance / num_runs);

    // Verify result
    CUDA_CHECK(hipMemcpy(h_dst, d_dst, size, hipMemcpyDeviceToHost));

    bool correct = true;
    int check_count = min(1000, N);

    for (int i = 0; i < check_count; i++)
    {
        if (h_dst[i] != h_src[i])
        {
            correct = false;
            printf("❌ Mismatch at %d: expected %.0f, got %.0f\n", i, h_src[i], h_dst[i]);
            break;
        }
    }

    if (correct && N > check_count)
    {
        for (int i = N - check_count; i < N; i++)
        {
            if (h_dst[i] != h_src[i])
            {
                correct = false;
                printf("❌ Mismatch at %d: expected %.0f, got %.0f\n", i, h_src[i], h_dst[i]);
                break;
            }
        }
    }

    if (correct)
    {
        printf("✅ TMA data verification passed! (checked %d elements)\n", min(2 * check_count, N));
    }

    printf("\n=== Grid-Stride TMA Results ===\n");
    printf("Data size: %.1f KB (%d floats)\n", size / 1024.0f, N);
    printf("Chunk size: %dKB (%d floats)\n", chunk_size_kb, elements_per_chunk);
    printf("Total chunks: %d, Blocks: %d\n", total_chunks, num_blocks);
    printf("Time (ms):\n");
    printf("  Min:    %.3f\n", min_time);
    printf("  Max:    %.3f\n", max_time);
    printf("  Avg:    %.3f ± %.3f\n", avg_time, std_dev);

    double bandwidth = (size * 2 / (avg_time / 1000.0)) / 1e9; // read+write
    printf("Bandwidth: %.1f GB/s\n", bandwidth);
    printf("Coefficient of Variation: %.2f%%\n", (std_dev / avg_time) * 100.0f);
    printf("✅ Using Hardware TMA (Tensor Memory Accelerator) with grid-stride loop\n");

    // Cleanup
    free(h_src);
    free(h_dst);
    CUDA_CHECK(hipFree(d_src));
    CUDA_CHECK(hipFree(d_dst));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    printf("\n=== Grid-Stride TMA Test Complete ===\n");
    return 0;
}
