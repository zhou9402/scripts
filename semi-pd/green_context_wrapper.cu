#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <pybind11/pybind11.h>

namespace py = pybind11;

// 全局变量存储 Green Context 状态
static CUgreenCtx g_green_ctx = nullptr;
static hipCtx_t g_context = nullptr;
static hipCtx_t g_default_context = nullptr;
static bool g_initialized = false;

// ==================== TMA Helper Functions ====================
__device__ __forceinline__ void fence_view_async_shared() {
    asm volatile("fence.proxy.async.shared::cta; \n" :: );
}

__device__ __forceinline__ void fence_barrier_init() {
    asm volatile("fence.mbarrier_init.release.cluster; \n" :: );
}

__device__ __forceinline__ void mbarrier_init(uint64_t* mbar_ptr, uint32_t arrive_count) {
    auto mbar_int_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(mbar_ptr));
    asm volatile("mbarrier.init.shared::cta.b64 [%1], %0;" :: "r"(arrive_count), "r"(mbar_int_ptr));
}

__device__ __forceinline__ void mbarrier_wait(uint64_t* mbar_ptr, uint32_t& phase) {
    auto mbar_int_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(mbar_ptr));
    asm volatile("{\n\t"
                 ".reg .pred       P1; \n\t"
                 "LAB_WAIT: \n\t"
                 "mbarrier.try_wait.parity.shared::cta.b64 P1, [%0], %1, %2; \n\t"
                 "@P1 bra DONE; \n\t"
                 "bra     LAB_WAIT; \n\t"
                 "DONE: \n\t"
                 "}" :: "r"(mbar_int_ptr), "r"(phase), "r"(0x989680));
    phase ^= 1;
}

__device__ __forceinline__ void mbarrier_arrive_and_expect_tx(uint64_t* mbar_ptr, int num_bytes) {
    auto mbar_int_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(mbar_ptr));
    asm volatile("mbarrier.arrive.expect_tx.shared::cta.b64 _, [%1], %0; \n\t" :: "r"(num_bytes), "r"(mbar_int_ptr));
}

constexpr uint64_t kEvictFirst = 0x12f0000000000000;
constexpr uint64_t kEvictNormal = 0x1000000000000000;

__device__ __forceinline__ void tma_load_1d(const void* smem_ptr, const void* gmem_ptr, uint64_t* mbar_ptr, int num_bytes,
                                            bool evict_first = true) {
    auto mbar_int_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(mbar_ptr));
    auto smem_int_ptr  = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
    const auto cache_hint = evict_first ? kEvictFirst : kEvictNormal;
    asm volatile("cp.async.bulk.shared::cluster.global.mbarrier::complete_tx::bytes.L2::cache_hint [%0], [%1], %2, [%3], %4;\n"
                 :: "r"(smem_int_ptr), "l"(gmem_ptr), "r"(num_bytes), "r"(mbar_int_ptr), "l"(cache_hint) : "memory");
}

__device__ __forceinline__ void tma_store_1d(const void* smem_ptr, const void* gmem_ptr, int num_bytes,
                                             bool evict_first = true) {
    auto smem_int_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
    const auto cache_hint = evict_first ? kEvictFirst : kEvictNormal;
    asm volatile("cp.async.bulk.global.shared::cta.bulk_group.L2::cache_hint [%0], [%1], %2, %3;\n"
                 :: "l"(gmem_ptr), "r"(smem_int_ptr), "r"(num_bytes), "l"(cache_hint) : "memory");
    asm volatile("cp.async.bulk.commit_group;");
}

template <int N = 0>
__device__ __forceinline__ void tma_store_wait() {
    asm volatile("cp.async.bulk.wait_group.read %0;" :: "n"(N) : "memory");
}

// ==================== GEMM Kernel ====================
// 高效的矩阵乘法kernel，使用shared memory和tile技术
// C = A * B, where A is MxK, B is KxN, C is MxN
template<int BLOCK_SIZE>
__global__ void gemm_kernel(const float* A, const float* B, float* C, 
                           int M, int N, int K) {
    // 每个线程块负责计算C中的一个tile
    int block_row = blockIdx.y;
    int block_col = blockIdx.x;
    
    // 线程在block中的位置
    int thread_row = threadIdx.y;
    int thread_col = threadIdx.x;
    
    // C中当前线程负责的元素位置
    int row = block_row * BLOCK_SIZE + thread_row;
    int col = block_col * BLOCK_SIZE + thread_col;
    
    // Shared memory for tiles of A and B
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    
    float c_value = 0.0f;
    
    // 计算需要多少个tile
    int num_tiles = (K + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    for (int tile = 0; tile < num_tiles; ++tile) {
        // 加载A的tile到shared memory
        int a_row = row;
        int a_col = tile * BLOCK_SIZE + thread_col;
        if (a_row < M && a_col < K) {
            As[thread_row][thread_col] = A[a_row * K + a_col];
        } else {
            As[thread_row][thread_col] = 0.0f;
        }
        
        // 加载B的tile到shared memory
        int b_row = tile * BLOCK_SIZE + thread_row;
        int b_col = col;
        if (b_row < K && b_col < N) {
            Bs[thread_row][thread_col] = B[b_row * N + b_col];
        } else {
            Bs[thread_row][thread_col] = 0.0f;
        }
        
        __syncthreads();
        
        // 计算部分乘积
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            c_value += As[thread_row][k] * Bs[k][thread_col];
        }
        
        __syncthreads();
    }
    
    // 写入结果
    if (row < M && col < N) {
        C[row * N + col] = c_value;
    }
}

// ==================== Compute Intensive Kernel ====================
// 计算密集型kernel：执行大量数学运算，数据加载量很小
__global__ void compute_intensive_kernel(float* input_params, float* output_results, int iterations_per_thread) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;
    
    // 从输入参数中读取基础参数（数据加载量很小）
    float base_val = input_params[0];
    float multiplier = input_params[1]; 
    float offset = input_params[2];
    float frequency = input_params[3];
    
    float result = 0.0f;
    float x = base_val + tid * 0.001f;  // 每个线程的起始值稍有不同
    
    // 执行大量计算密集型操作
    for (int i = 0; i < iterations_per_thread; i++) {
        // 复杂的数学计算：组合多种数学函数
        float temp1 = sinf(x * frequency + i * 0.01f);
        float temp2 = cosf(x * multiplier + i * 0.02f);
        float temp3 = expf(-x * 0.1f + i * 0.001f);
        float temp4 = logf(fabsf(x) + 1.0f + i * 0.001f);
        
        // 更多计算操作
        float temp5 = powf(fabsf(temp1), 0.3f);
        float temp6 = sqrtf(fabsf(temp2 * temp3) + 1.0f);
        float temp7 = tanhf(temp4 * 0.1f);
        
        // 组合计算结果
        result += temp1 * temp2 + temp3 * temp4 + temp5 * temp6 + temp7;
        
        // 更新x值进行下一轮计算
        x += 0.001f;
        
        // 额外的计算操作增加计算密度
        float temp8 = sinf(result * 0.01f) * cosf(x * 0.02f);
        float temp9 = expf(-fabsf(temp8) * 0.1f);
        result = result * 0.99f + temp9 * 0.01f;
    }
    
    // 写入最终结果
    if (tid < total_threads) {
        output_results[tid] = result;
    }
}

// ==================== TMA Load Kernel ====================
// TMA Grid-stride Load Kernel with 32KB chunks
__global__ void tma_load_kernel(float4* src_data, float4* dst_data, int total_elements) {
    // 每次copy 32KB = 8192个float = 2048个float4
    const int elements_per_chunk = 2048;  // 32KB / 16 bytes per float4
    const int bytes_per_chunk = elements_per_chunk * 16;  // 32KB
    
    extern __shared__ __align__(1024) uint8_t smem_buffer[];
    
    // TMA barrier for load
    __shared__ uint64_t tma_mbarrier;
    uint32_t tma_phase = 0;

    if (threadIdx.x == 0) {
        mbarrier_init(&tma_mbarrier, 1);
        fence_view_async_shared();
        fence_barrier_init();
    }
    __syncthreads();

    // Grid-stride loop: 每个block轮流处理32KB数据块
    for (int chunk_start = blockIdx.x * elements_per_chunk; 
         chunk_start < total_elements; 
         chunk_start += gridDim.x * elements_per_chunk) {
        
        // 计算当前chunk的实际大小
        int chunk_end = min(chunk_start + elements_per_chunk, total_elements);
        int elements_to_process = chunk_end - chunk_start;
        int bytes_to_process = elements_to_process * 16;  // float4 = 16 bytes
        
        // 跳过空的chunk
        if (elements_to_process <= 0) {
            break;
        }

        // --- TMA Load: Global -> Shared ---
        if (threadIdx.x == 0)
        {
            // 计算当前chunk的源地址和目标地址
            const float4* chunk_src = src_data + chunk_start;
            float4* chunk_dst = dst_data + chunk_start;
            
            // TMA加载：从global memory的当前chunk位置加载到shared memory
            tma_load_1d(smem_buffer, chunk_src, &tma_mbarrier, bytes_to_process);
            mbarrier_arrive_and_expect_tx(&tma_mbarrier, bytes_to_process);
            
            // 等待TMA加载完成
            mbarrier_wait(&tma_mbarrier, tma_phase);
            
            // --- TMA Store: Shared -> Global ---
            // TMA存储：从shared memory存储到global memory的当前chunk位置
            tma_store_1d(smem_buffer, chunk_dst, bytes_to_process);
            tma_store_wait();
        }
        
        // 等待当前chunk的TMA操作完成再进行下一个chunk
        __syncthreads();
    }
}

bool create_green_context(int sm_count = 8) {
    if (g_initialized) {
        return true; // 已经初始化
    }
    
    // 保存当前的默认context
    hipError_t res = hipCtxGetCurrent(&g_default_context);
    if (res != hipSuccess) {
        std::cerr << "无法获取当前默认context" << std::endl;
        return false;
    }
    
    hipDevice_t device;
    CUdevResource dev_resource = {};
    CUdevResource sm_resources[2] = {{}, {}};
    CUdevResourceDesc desc = nullptr;
    unsigned int flags = CU_GREEN_CTX_DEFAULT_STREAM;
    unsigned int split_count = 1;
    unsigned int min_sm_count = sm_count;
    
    // 初始化 CUDA Driver
    res = hipInit(0);
    if (res != hipSuccess) {
        std::cerr << "CUDA初始化失败" << std::endl;
        return false;
    }
    
    // 获取设备
    res = hipDeviceGet(&device, 0);
    if (res != hipSuccess) {
        std::cerr << "获取设备失败" << std::endl;
        return false;
    }
    
    // 获取设备的 SM 资源
    res = cuDeviceGetDevResource(device, &dev_resource, CU_DEV_RESOURCE_TYPE_SM);
    if (res != hipSuccess) {
        std::cerr << "获取设备资源失败" << std::endl;
        return false;
    }
    
    // 分割 SM 资源
    res = cuDevSmResourceSplitByCount(&sm_resources[0], &split_count, 
                                      &dev_resource, &sm_resources[1], 
                                      0, min_sm_count);
    if (res != hipSuccess) {
        std::cerr << "分割 SM 资源失败" << std::endl;
        return false;
    }
    
    // 生成资源描述符
    res = cuDevResourceGenerateDesc(&desc, &sm_resources[0], 1);
    if (res != hipSuccess) {
        std::cerr << "生成资源描述符失败" << std::endl;
        return false;
    }
    
    // 创建 Green Context
    res = cuGreenCtxCreate(&g_green_ctx, desc, device, flags);
    if (res != hipSuccess) {
        std::cerr << "创建Green Context失败" << std::endl;
        return false;
    }
    
    // 转换为普通 context
    res = cuCtxFromGreenCtx(&g_context, g_green_ctx);
    if (res != hipSuccess) {
        std::cerr << "转换 Green Context 失败" << std::endl;
        return false;
    }
    
    g_initialized = true;
    return true;
}

void destroy_green_context() {
    if (g_initialized && g_green_ctx) {
        // 先切换回默认context
        if (g_default_context) {
            hipCtxSetCurrent(g_default_context);
        }
        cuGreenCtxDestroy(g_green_ctx);
        g_green_ctx = nullptr;
        g_context = nullptr;
        g_default_context = nullptr;
        g_initialized = false;
    }
}

bool switch_to_green_context() {
    if (!g_initialized || !g_context) {
        std::cerr << "Green Context未初始化，请先调用create_green_context" << std::endl;
        return false;
    }
    
    hipError_t res = hipCtxSetCurrent(g_context);
    if (res != hipSuccess) {
        std::cerr << "切换到Green Context失败" << std::endl;
        return false;
    }
    
    return true;
}

bool switch_to_default_context() {
    if (!g_default_context) {
        std::cerr << "默认Context不可用" << std::endl;
        return false;
    }
    
    hipError_t res = hipCtxSetCurrent(g_default_context);
    if (res != hipSuccess) {
        std::cerr << "切换到默认Context失败" << std::endl;
        return false;
    }
    
    return true;
}

bool is_green_context_active() {
    return g_initialized;
}

// 检查GPU是否支持TMA (Hopper架构)
bool check_tma_support() {
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    
    return prop.major >= 9;  // Hopper架构 (compute_90+)
}

// GEMM Kernel接口 - 矩阵乘法 C = A * B
void run_gemm_kernel(torch::Tensor A, torch::Tensor B, torch::Tensor C) {
    // 基本检查
    TORCH_CHECK(A.device().is_cuda(), "Tensor A must be on CUDA");
    TORCH_CHECK(B.device().is_cuda(), "Tensor B must be on CUDA");  
    TORCH_CHECK(C.device().is_cuda(), "Tensor C must be on CUDA");
    TORCH_CHECK(A.dtype() == torch::kFloat32, "Tensor A must be float32");
    TORCH_CHECK(B.dtype() == torch::kFloat32, "Tensor B must be float32");
    TORCH_CHECK(C.dtype() == torch::kFloat32, "Tensor C must be float32");
    TORCH_CHECK(A.is_contiguous(), "Tensor A must be contiguous");
    TORCH_CHECK(B.is_contiguous(), "Tensor B must be contiguous");
    TORCH_CHECK(C.is_contiguous(), "Tensor C must be contiguous");
    
    // 获取矩阵维度
    TORCH_CHECK(A.dim() == 2, "Tensor A must be 2D");
    TORCH_CHECK(B.dim() == 2, "Tensor B must be 2D");
    TORCH_CHECK(C.dim() == 2, "Tensor C must be 2D");
    
    int M = A.size(0);
    int K = A.size(1);
    int K_B = B.size(0);
    int N = B.size(1);
    int M_C = C.size(0);
    int N_C = C.size(1);
    
    TORCH_CHECK(K == K_B, "A.cols must equal B.rows");
    TORCH_CHECK(M == M_C, "A.rows must equal C.rows");
    TORCH_CHECK(N == N_C, "B.cols must equal C.cols");
    
    // 获取数据指针
    float* A_ptr = A.data_ptr<float>();
    float* B_ptr = B.data_ptr<float>();
    float* C_ptr = C.data_ptr<float>();
    
    // 设置kernel配置
    const int BLOCK_SIZE = 16;  // 16x16 tile size
    dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_dim((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    // 调用GEMM kernel
    gemm_kernel<BLOCK_SIZE><<<grid_dim, block_dim>>>(A_ptr, B_ptr, C_ptr, M, N, K);
    
    // 同步并检查错误
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::runtime_error("GEMM kernel failed: " + std::string(hipGetErrorString(error)));
    }
}

// Compute Intensive Kernel接口 - 高计算密度，低数据传输
void run_compute_intensive_kernel(torch::Tensor input_params, int iterations_per_thread) {
    // 基本检查
    TORCH_CHECK(input_params.device().is_cuda(), "Input params must be on CUDA");
    TORCH_CHECK(input_params.dtype() == torch::kFloat32, "Input params must be float32");
    TORCH_CHECK(input_params.is_contiguous(), "Input params must be contiguous");
    TORCH_CHECK(input_params.numel() >= 4, "Need at least 4 input parameters");
    
    // 获取输入参数指针
    float* input_data = input_params.data_ptr<float>();
    
    // 设置kernel配置 - 使用大量线程进行计算
    int threads_per_block = 256;
    int num_blocks = 64;  // 可以根据需要调整
    int total_threads = num_blocks * threads_per_block;
    
    // 创建输出tensor存储每个线程的计算结果
    torch::Tensor output_results = torch::zeros({total_threads}, 
                                               torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));
    float* output_data = output_results.data_ptr<float>();
    
    // 调用计算密集型kernel
    compute_intensive_kernel<<<num_blocks, threads_per_block>>>(
        input_data, output_data, iterations_per_thread);
    
    // 同步等待kernel完成
    hipDeviceSynchronize();
    
    // 检查是否有错误
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::runtime_error("Compute intensive kernel failed: " + std::string(hipGetErrorString(error)));
    }
}

// TMA Load Kernel接口 - 需要Hopper架构
void run_tma_load_kernel(torch::Tensor src_tensor) {
    // 基本检查
    TORCH_CHECK(src_tensor.device().is_cuda(), "Tensor must be on CUDA");
    TORCH_CHECK(src_tensor.dtype() == torch::kFloat32, "Tensor must be float32");
    TORCH_CHECK(src_tensor.is_contiguous(), "Tensor must be contiguous");
    
    int tensor_elements = src_tensor.numel();
    TORCH_CHECK(tensor_elements >= 8, "Tensor too small, need at least 8 elements");
    
    // 获取源数据指针
    float4* src_data = reinterpret_cast<float4*>(src_tensor.data_ptr<float>());
    
    // 创建目标tensor
    torch::Tensor dst_tensor = torch::empty_like(src_tensor);
    float4* dst_data = reinterpret_cast<float4*>(dst_tensor.data_ptr<float>());
    
    // TMA kernel配置 - 32KB chunks, grid-stride loop
    int num_float4 = tensor_elements / 4;
    const int elements_per_chunk = 2048;  // 32KB / 16 bytes per float4
    
    // 计算需要的总chunk数量
    int total_chunks = (num_float4 + elements_per_chunk - 1) / elements_per_chunk;
    
    // 设置block数量：使用grid-stride pattern
    int threads_per_block = 1024;  // TMA需要足够的线程来占用SM
    
    // 设置shared memory大小 - 32KB per block
    size_t smem_size = elements_per_chunk * 16;  // 32KB
    
    // 设置kernel属性
    hipFuncSetAttribute(reinterpret_cast<const void*>(tma_load_kernel), 
                        hipFuncAttributeMaxDynamicSharedMemorySize, 
                        smem_size);
    
    // 调用TMA kernel
    tma_load_kernel<<<total_chunks, threads_per_block, smem_size>>>(src_data, dst_data, num_float4);
}

// Python 绑定
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("create_green_context", &create_green_context, 
          "Create Green Context with limited SM count", 
          py::arg("sm_count") = 8);
    
    m.def("destroy_green_context", &destroy_green_context, 
          "Destroy Green Context");
    
    m.def("switch_to_green_context", &switch_to_green_context, 
          "Switch to Green Context");
    
    m.def("switch_to_default_context", &switch_to_default_context, 
          "Switch to default Context");
    
    m.def("is_green_context_active", &is_green_context_active, 
          "Check if Green Context is active");
    
    m.def("check_tma_support", &check_tma_support, 
          "Check if current GPU supports TMA (Hopper architecture)");
    
    m.def("run_tma_load_kernel", &run_tma_load_kernel, 
          "Run TMA load kernel with grid-stride loop (32KB chunks, requires Hopper)",
          py::arg("src_tensor"));

    m.def("run_compute_intensive_kernel", &run_compute_intensive_kernel, 
          "Run compute intensive kernel (high computation density, low data transfer)",
          py::arg("input_params"), py::arg("iterations_per_thread"));

    m.def("run_gemm_kernel", &run_gemm_kernel, 
          "Run GEMM kernel (matrix multiplication)",
          py::arg("A"), py::arg("B"), py::arg("C"));
} 